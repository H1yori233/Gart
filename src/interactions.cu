#include "hip/hip_runtime.h"
#include "interactions.h"
#include "random.h"
#include "texture.h"

__host__ __device__ glm::vec3 calculateRandomDirectionInHemisphere(
    glm::vec3 normal,
    thrust::default_random_engine &rng)
{
    thrust::uniform_real_distribution<float> u01(0, 1);

    float up = sqrt(u01(rng));      // cos(theta)
    float over = sqrt(1 - up * up); // sin(theta)
    float around = u01(rng) * TWO_PI;

    // Find a direction that is not the normal based off of whether or not the
    // normal's components are all equal to sqrt(1/3) or whether or not at
    // least one component is less than sqrt(1/3). Learned this trick from
    // Peter Kutz.

    glm::vec3 directionNotNormal;
    if (abs(normal.x) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(1, 0, 0);
    }
    else if (abs(normal.y) < SQRT_OF_ONE_THIRD)
    {
        directionNotNormal = glm::vec3(0, 1, 0);
    }
    else
    {
        directionNotNormal = glm::vec3(0, 0, 1);
    }

    // Use not-normal direction to generate two perpendicular directions
    glm::vec3 perpendicularDirection1 =
        glm::normalize(glm::cross(normal, directionNotNormal));
    glm::vec3 perpendicularDirection2 =
        glm::normalize(glm::cross(normal, perpendicularDirection1));

    return up * normal + 
           cos(around) * over * perpendicularDirection1 + 
           sin(around) * over * perpendicularDirection2;
}

// Heavily Ref: https://github.com/cs87-dartmouth/darts-2024/blob/main/src/materials/material.cpp
__host__ __device__ float fresnelDielectric(float cos_theta_i, 
    float eta_i, float eta_t)
{
    // Using Sahl-Snell's law, calculate the squared sine of the angle between the normal and the transmitted ray
    float eta          = eta_i / eta_t;
    float sin_theta_t2 = eta * eta * (1 - cos_theta_i * cos_theta_i);

    // Total internal reflection!
    if (sin_theta_t2 > 1.0f)
        return 1.0f;

    float cos_theta_t = sqrtf(1.0f - sin_theta_t2);

    float Rs = (eta_i * cos_theta_i - eta_t * cos_theta_t) / 
               (eta_i * cos_theta_i + eta_t * cos_theta_t);
    float Rp = (eta_t * cos_theta_i - eta_i * cos_theta_t) / 
               (eta_t * cos_theta_i + eta_i * cos_theta_t);

    return 0.5f * (Rs * Rs + Rp * Rp);
}

__host__ __device__ void scatterRay(
    PathSegment &pathSegment,
    glm::vec3 intersect,
    glm::vec3 normal,
    const Material &m,
    const glm::vec2 &uv,
    const DevTexturePool &texture_pool,
    thrust::default_random_engine &rng)
{
    // TODO: implement this.
    // A basic implementation of pure-diffuse shading will just call the
    // calculateRandomDirectionInHemisphere defined above.
    thrust::uniform_real_distribution<float> u01(0, 1);

    if (m.hasReflective > 0.f)
    {
        auto wi = pathSegment.ray.direction;
        auto wo = glm::reflect(wi, normal);
        wo = glm::normalize(wo);

        pathSegment.ray.origin = intersect + EPSILON * wo;
        pathSegment.ray.direction = wo;
        pathSegment.color *= eval(m.color, uv, 0.0f, texture_pool);
    }
    else if (m.hasRefractive > 0.f)
    {
        auto wi = pathSegment.ray.direction;
        float cos_theta_i = -glm::dot(wi, normal);
        float sin_theta_i = sqrtf(1 - cos_theta_i * cos_theta_i);

        float ior, refl;
        if (cos_theta_i > 0)
        {
            ior = 1 / m.indexOfRefraction;
            refl = fresnelDielectric(cos_theta_i, 1.f, m.indexOfRefraction);
        }
        else
        {
            ior = m.indexOfRefraction;
            refl = fresnelDielectric(cos_theta_i, m.indexOfRefraction, 1.f);
        }

        // Schlick's approximation
        // float r0 = (1 - ior) / (1 + ior);
        // r0 = r0 * r0;
        // float refl = r0 + (1 - r0) * std::pow((1 - cos_theta_i), 5);
        
        auto wo = wi;
        if ((ior * sin_theta_i > 1) || (u01(rng) < refl)) 
        {
            wo = glm::reflect(wi, normal);
        }
        else 
        {
            wo = glm::refract(wi, normal, ior);
        }
        wo = glm::normalize(wo);
        
        // Very Strange, EPSILON = 0.00001f wont work
        // only 0.0001f works.
        pathSegment.ray.origin = intersect + 0.0001f * wo;
        pathSegment.ray.direction = wo;
        pathSegment.color *= eval(m.color, uv, 0.0f, texture_pool);
    }
    else
    {
        // auto wo = calculateRandomDirectionInHemisphere(normal, rng);
        // wo = glm::normalize(wo); 
        auto lo     = randomHemisphereCosine(u01(rng), u01(rng));

        auto onb    = ONB(normal);
        auto wo     = glm::normalize(onb.localToWorld(lo));

        pathSegment.ray.origin = intersect + EPSILON * wo;
        pathSegment.ray.direction = wo;

        // * pdf and then / pdf, so ignore it
        pathSegment.color *= eval(m.color, uv, 0.0f, texture_pool);
    }

    pathSegment.remainingBounces--;
}
